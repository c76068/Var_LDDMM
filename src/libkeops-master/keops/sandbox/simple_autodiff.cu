#include "hip/hip_runtime.h"
// nvcc -I.. -Wno-deprecated-gpu-targets -DCUDA_BLOCK_SIZE=192 -D__TYPE__=float -std=c++11 -Xcompiler -fPIC -shared -o build/simple_autodiff.so simple_autodiff.cu

#include "core/formulas/constants.h"
#include "core/formulas/maths.h"
#include "core/formulas/kernels.h"
#include "core/formulas/norms.h"
#include "core/formulas/factorize.h"

#include "core/GpuConv2D.cu"

using namespace keops;


// define variables
using X = Var<1,3>; 	// X is the second variable and represents a 3D vector
using Y = Var<2,3>; 	// Y is the third variable and represents a 3D vector
using U = Var<3,4>; 	// U is the fourth variable and represents a 4D vector
using V = Var<4,4>; 	// V is the fifth variable and represents a 4D vector
using Beta = Var<5,3>;	// Beta is the sixth variable and represents a 3D vector
using C = Param<0,1>;	// C is the first variable and is a scalar parameter

// define F = <U,V>^2 * exp(-C*|X-Y|^2) * Beta in usual notations
using F = Scal<Square<Scalprod<U,V>>,Scal<Exp<Scal<C,Minus<SqNorm2<Subtract<X,Y>>>>>,Beta>>;

using FUNCONVF = typename Generic<F>::sEval;

extern "C" int FConv(float ooSigma2, float* x, float* y, float* u, float* v, float* beta, float* gamma, int nx, int ny) {
    float params[1];
    params[0] = ooSigma2;
    return GpuConv2D(FUNCONVF(), nx, ny, gamma, params, x, y, u, v, beta);
}


// now define the gradient wrt XX
using Eta = Var<6,F::DIM>;	// new variable is in seventh position and is input of gradient
using GX = Grad<F,X,Eta>;

using FUNCONVGX = typename Generic<GX>::sEval;

extern "C" int GXConv(float ooSigma2, float* x, float* y, float* u, float* v, float* beta, float* eta, float* gamma, int nx, int ny) {
    float params[1];
    params[0] = ooSigma2;
    return GpuConv2D(FUNCONVGX(), nx, ny, gamma, params, x, y, u, v, beta, eta);
}


// now define the gradient wrt Y. 
using GY = Grad<F,Y,Eta>;

// since Y is a j variable, all i variables become j variables and conversely : this is why we put 1 as second template argument after GY :
using FUNCONVGY = typename Generic<GY,1>::sEval;

extern "C" int GYConv(float ooSigma2, float* x, float* y, float* u, float* v, float* beta, float* eta, float* gamma, int nx, int ny) {
    float params[1];
    params[0] = ooSigma2;
    return GpuConv2D(FUNCONVGY(), ny, nx, params, gamma, x, y, u, v, beta, eta);
}



