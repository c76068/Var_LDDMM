#include "hip/hip_runtime.h"
// test convolution with autodiff
// compile with
//		nvcc -I.. -Wno-deprecated-gpu-targets -std=c++11 -O2 -o build/test_autodiff test_autodiff.cu

// we define an arbitrary function using available blocks,
// then test its convolution on the GPU, then get its gradient and test again the convolution

// Here we build the function F(x,y,u,v,beta) = <u,v>^2 * exp(-C*|x-y|^2) * beta
// where x, y, beta are 3D vectors, and u, v are 4D vectors
// and the convolution is gamma_i = sum_j F(x_i,y_j,u_i,v_j,beta_j)
// then we define G(x,y,u,v,beta,eta) = gradient of F with respect to x, with new input variable eta (3D)
// and the new convolution is gamma_i = sum_j G(x_i,y_j,u_i,v_j,beta_j,eta_i)

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ctime>
#include <algorithm>

#include "core/formulas/constants.h"
#include "core/formulas/maths.h"
#include "core/formulas/kernels.h"
#include "core/formulas/norms.h"
#include "core/formulas/factorize.h"

#include "core/GpuConv1D.cu"
#include "core/GpuConv2D.cu"
#include "core/CpuConv.cpp"

using namespace keops;

__TYPE__ floatrand() {
    return ((__TYPE__) std::rand())/RAND_MAX-.5;    // random value between -.5 and .5
}

template < class V > void fillrandom(V& v) {
    generate(v.begin(), v.end(), floatrand);    // fills vector with random values
}

int main() {
    // In this part we define the symbolic variables of the function
    using X = Var<1,3,0>; 	// X is the second variable and represents a 3D vector
    using Y = Var<2,3,1>; 	// Y is the third variable and represents a 3D vector
    using U = Var<3,4,0>; 	// U is the fourth variable and represents a 4D vector
    using V = Var<4,4,1>; 	// V is the fifth variable and represents a 4D vector
    using Beta = Var<5,3,1>;	// Beta is the sixth variable and represents a 3D vector
    using C = Param<0,1>;		// C is the first variable and is a scalar parameter

    // symbolic expression of the function ------------------------------------------------------

    // here we define F = <U,V>^2 * exp(-C*|X-Y|^2) * Beta in usual notations
    using F = Scal<Square<Scalprod<U,V>>, Scal<Exp<Scal<C,Minus<SqNorm2<Subtract<X,Y>>>>>,Beta>>;

    using FUNCONVF = typename Generic<F>::sEval;

    // gradient with respect to X ---------------------------------------------------------------
    using Eta = Var<6,F::DIM,0>; // new variable is in seventh position and is input of gradient
    using GX = Grad<F,X,Eta>;

    /*
     * Using GX = Grad<F,X,Eta> = (\partial_X F).Eta in a convolution sum (Generic<...>) makes sense.
     * Indeed, we know that
     *
     *      FUNCONVF_i = \sum_j F( P, X^0_i, X^1_i, ..., Y^0_j, Y^1_j, ... ).
     *
     * Then, since FUNCONVF_i only depends on the i-th line of X^n,
     *
     * (\partial_{X^n} FUNCONVF).Eta = \sum_i (\partial_{X^n  } FUNCONVF_i).Eta_i       (definition of the L2 scalar product)
     *
     *                                        | 0 0 ................................. 0 |
     *                                        | 0 0 ................................. 0 |
     *                               = \sum_i |  (\partial_{X^n_i} FUNCONVF_i).Eta_i    | <- (on the i-th line).
     *                                        | 0 0 ................................. 0 |
     *                                        | 0 0 ................................. 0 |
     *                                        | 0 0 ................................. 0 |
     *
     *                                        |  (\partial_{X^n_0} FUNCONVF_0).Eta_0    |
     *                                        |  (\partial_{X^n_1} FUNCONVF_1).Eta_1    |
     *                               =        |                    .                    |
     *                                        |                    .                    |
     *                                        |                    .                    |
     *                                        |  (\partial_{X^n_I} FUNCONVF_I).Eta_I    |
     *
     * But then, by linearity of the gradient operator,
     *
     * (\partial_{X^n_i} FUNCONVF_i).Eta_i = \sum_j (\partial_{X^n} F( P, X^0_i, ..., Y^0_j, ... )).Eta_i
     *
     * (\partial_{X^n} FUNCONVF).Eta is therefore equal to the "generic kernel product" with
     * summation on j, with the summation term being
     *
     *    (\partial_{X^n_i} F( P, X^0_i, ..., Y^0_j, ... )).Eta_i  = Grad<F,X^n,Eta>
     *
     */
    using FUNCONVGX = typename Generic<GX>::sEval;

    // gradient with respect to Y  --------------------------------------------------------------
    using GY = Grad<F,Y,Eta>;

    /*
     * Using GY = Grad<F,Y,Eta> = (\partial_Y F).Eta in a convolution sum (Generic<...>) makes sense...
     * IF YOU CHANGE THE SUMMATION VARIABLE FROM j TO i !
     * Indeed, we know that
     *
     *      FUNCONVF_i = \sum_j F( P, X^0_i, X^1_i, ..., Y^0_j, Y^1_j, ... ).
     *
     * Hence, doing the computations :
     *
     * (\partial_{Y^m} FUNCONVF).Eta
     *    = \sum_i    (\partial_{Y^m  } FUNCONVF_i).Eta_i                          (definition of the L2 scalar product)
     *    = \sum_i    (\partial_{Y^m  } \sum_j F(P,X^0_i, ...,Y^0_j,...) ).Eta_i   (FUNCONVF_i = ...)
     *    = \sum_j    \sum_i (\partial_{Y^m  } F(P,X^0_i, ...,Y^0_j,...) ).Eta_i   (Fubini theorem + linearity of \partial_{Y^M})
     *
     *              | 0 0 .................................................... 0 | (the summation term only depends on Y^m_j)
     *              | 0 0 .................................................... 0 |
     *    = \sum_j  | \sum_i (\partial_{Y^m_j} F(P,X^0_i, ...,Y^0_j,...) ).Eta_i | <- (on the j-th line)
     *              | 0 0 .................................................... 0 |
     *              | 0 0 .................................................... 0 |
     *              | 0 0 .................................................... 0 |
     *              | 0 0 .................................................... 0 |
     *
     *              | \sum_i (\partial_{Y^m_0} F(P,X^0_i, ...,Y^0_0,...) ).Eta_i |
     *              | \sum_i (\partial_{Y^m_1} F(P,X^0_i, ...,Y^0_1,...) ).Eta_i |
     *    =         |                               .                            |
     *              |                               .                            |
     *              |                               .                            |
     *              |                               .                            |
     *              | \sum_i (\partial_{Y^m_J} F(P,X^0_i, ...,Y^0_J,...) ).Eta_i |
     *
     *
     * (\partial_{Y^m} FUNCONVF).Eta is therefore equal to the "generic kernel product" with
     * summation on i (and not j !), with the summation term being
     *
     *    (\partial_{Y^m_j} F( P, X^0_i, ..., Y^0_j, ... )).Eta_i  = Grad<F,Y^m,Eta>
     *
     */
    // parameter 1 after GY means i and j variables must be swapped,
    // i.e. we do a summation on "i" using a code which is hardcoded for summation wrt. "j" :
    using FUNCONVGY = typename Generic<GY,1>::sEval;

    // now we test ------------------------------------------------------------------------------

    int Nx=5000, Ny=2000;

    std::vector<__TYPE__> vf(Nx*F::DIM);    fillrandom(vf); __TYPE__ *f = vf.data();
    std::vector<__TYPE__> vx(Nx*X::DIM);    fillrandom(vx); __TYPE__ *x = vx.data();
    std::vector<__TYPE__> vy(Ny*Y::DIM);    fillrandom(vy); __TYPE__ *y = vy.data();
    std::vector<__TYPE__> vu(Nx*U::DIM);    fillrandom(vu); __TYPE__ *u = vu.data();
    std::vector<__TYPE__> vv(Ny*V::DIM);    fillrandom(vv); __TYPE__ *v = vv.data();
    std::vector<__TYPE__> vb(Ny*Beta::DIM); fillrandom(vb); __TYPE__ *b = vb.data();

    std::vector<__TYPE__> resgpu2D(Nx*F::DIM), resgpu1D(Nx*F::DIM), rescpu(Nx*F::DIM);

    __TYPE__ params[1];
    __TYPE__ Sigma = 4.0;
    params[0] = 1.0/(Sigma*Sigma);

    clock_t begin, end;

    begin = clock();
    int deviceID = 1;
    hipSetDevice(deviceID);
    end = clock();
    std::cout << "time for GPU initialization : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    std::cout << "blank run" << std::endl;
    begin = clock();
    GpuConv2D(FUNCONVF(), Nx, Ny, f, params, x, y, u, v, b);
    end = clock();
    std::cout << "time for blank run : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    std::cout << "testing function F" << std::endl;
    begin = clock();
    GpuConv2D(FUNCONVF(), Nx, Ny, f, params, x, y, u, v, b);
    end = clock();
    std::cout << "time for GPU computation (2D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu2D = vf;

    begin = clock();
    GpuConv1D(FUNCONVF(), Nx, Ny, f, params, x, y, u, v, b);
    end = clock();
    std::cout << "time for GPU computation (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu1D = vf;

    begin = clock();
    CpuConv(FUNCONVF(), Nx, Ny, f, params, x, y, u, v, b);
    end = clock();
    std::cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    rescpu = vf;

    // display mean of errors
    __TYPE__ s = 0;
    for(int i=0; i<Nx*F::DIM; i++)
        s += std::abs(resgpu2D[i]-rescpu[i]);
    std::cout << "mean abs error 2D =" << s/Nx << std::endl;

    s = 0;
    for(int i=0; i<Nx*F::DIM; i++)
        s += std::abs(resgpu1D[i]-rescpu[i]);
    std::cout << "mean abs error 1D =" << s/Nx << std::endl;




    std::vector<__TYPE__> ve(Nx*Eta::DIM); fillrandom(ve); __TYPE__ *e = ve.data();

    std::cout << "testing function GX" << std::endl;
    begin = clock();
    GpuConv2D(FUNCONVGX(), Nx, Ny, f, params, x, y, u, v, b, e);
    end = clock();
    std::cout << "time for GPU computation (2D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu2D = vf;

    begin = clock();
    GpuConv1D(FUNCONVGX(), Nx, Ny, f, params, x, y, u, v, b, e);
    end = clock();
    std::cout << "time for GPU computation (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu1D = vf;

    begin = clock();
    CpuConv(FUNCONVGX(), Nx, Ny, f, params, x, y, u, v, b, e);
    end = clock();
    std::cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    rescpu = vf;

    // display mean of errors
    s = 0;
    for(int i=0; i<Nx*GX::DIM; i++)
        s += std::abs(resgpu2D[i]-rescpu[i]);
    std::cout << "mean abs error 2D =" << s/Nx << std::endl;

    s = 0;
    for(int i=0; i<Nx*GX::DIM; i++)
        s += std::abs(resgpu1D[i]-rescpu[i]);
    std::cout << "mean abs error 1D =" << s/Nx << std::endl;



    // gradient wrt Y, which is a "j" variable.

    rescpu.resize(Ny*GY::DIM);
    resgpu2D.resize(Ny*GY::DIM);
    resgpu1D.resize(Ny*GY::DIM);
    vf.resize(Ny*GY::DIM);
    f = vf.data();

    std::cout << "testing function GY" << std::endl;
    begin = clock();
    GpuConv2D(FUNCONVGY(), Ny, Nx, f, params, x, y, u, v, b, e);
    end = clock();
    std::cout << "time for GPU computation (2D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu2D = vf;

    begin = clock();
    GpuConv1D(FUNCONVGY(), Ny, Nx, f, params, x, y, u, v, b, e);
    end = clock();
    std::cout << "time for GPU computation (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    resgpu1D = vf;

    begin = clock();
    CpuConv(FUNCONVGY(), Ny, Nx, f, params, x, y, u, v, b, e);
    end = clock();
    std::cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    rescpu = vf;

    // display mean of errors
    s = 0;
    for(int i=0; i<Ny*GY::DIM; i++)
        s += std::abs(resgpu2D[i]-rescpu[i]);
    std::cout << "mean abs error 2D=" << s/Ny << std::endl;

    s = 0;
    for(int i=0; i<Ny*GY::DIM; i++)
        s += std::abs(resgpu1D[i]-rescpu[i]);
    std::cout << "mean abs error 1D=" << s/Ny << std::endl;



}



