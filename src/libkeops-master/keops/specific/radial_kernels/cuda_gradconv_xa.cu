/*
*	This cuda routine allows one to compute the derivative wrt the dual vector 'a' of the derivative
*	wrt 'x' of the expression
*		K(x_i,y_j) @ b_j =  sum_j f( |x_i-y_j|^2 ) b_j
*
*
*	We're looking for the gradient with respect to a of
*
*	< e, K(s,a,x,y,b) >  =  \sum_{i,j} f_s'( |x_i-y_j|^2 ) * < a_i, b_j > * 2 < e_i, x_i-y_j>,
*
*	which is an N-by-E array g_i (i from 1 to N), where each line is equal to
*
*	g_i  =  \sum_j 2* f_s'( |x_i-y_j|^2 ) * < e_i, x_i-y_j> * b_j
*
*	We will compute this sum over the index 'j' on the GPU, with 'one thread' = 'one index i'.
*	Data will be stored as follow:
*	  - e_i in the thread memory
*	  - x_i in the thread memory
*	  - y_j in the SharedData
*	  - b_j in the SharedData (beta_j, really)
*
*
* Author : Jean Feydy, heavily based on the work of Joan Glaunès and Benjamin Charlier.
*
*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "specific/radial_kernels/radial_kernels.h"
#include "specific/radial_kernels/cuda_gradconv_xa.cx"


//////////////////////////////////////////////////////
/////////// CPU -> GPU -> CPU routines ///////////////
//////////////////////////////////////////////////////


template <typename TYPE, KernelFun KernelFp >
int KernelGpuGradConvXA(TYPE ooSigma2,               // 1 / sigma^2
                        TYPE* e_h,                     // N-by-D array (same as x)
                        TYPE* alpha_h, TYPE* x_h,     // N-by-E, N-by-D arrays
                        TYPE* y_h,     TYPE* beta_h,  // M-by-D, M-by-E arrays
                        TYPE* gamma_h,                 // Output: N-by-E (same as alpha)
                        int dimPoint, int dimVect, int nx, int ny) { // D, E, N, M

    // Data on the device.
    TYPE* e_d;
    TYPE* alpha_d;
    TYPE* x_d;
    TYPE* y_d;
    TYPE* beta_d;
    TYPE* gamma_d;

    // Allocate arrays on device.
    hipMalloc((void**)&e_d,     sizeof(TYPE)*(nx*dimPoint));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*dimVect ));
    hipMalloc((void**)&x_d,     sizeof(TYPE)*(nx*dimPoint));
    hipMalloc((void**)&y_d,     sizeof(TYPE)*(ny*dimPoint));
    hipMalloc((void**)&beta_d,  sizeof(TYPE)*(ny*dimVect ));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*dimVect )); // Output: N-by-E (same as alpha)

    // Send data from host to device.
    hipMemcpy(e_d,     e_h,     sizeof(TYPE)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*dimVect ), hipMemcpyHostToDevice);
    hipMemcpy(x_d,     x_h,     sizeof(TYPE)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(y_d,     y_h,     sizeof(TYPE)*(ny*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(beta_d,  beta_h,  sizeof(TYPE)*(ny*dimVect ), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    // Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time :
    if(     dimPoint==1 && dimVect==1)
        KernelGpuGradConvXAOnDevice<TYPE,1,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==1)
        KernelGpuGradConvXAOnDevice<TYPE,2,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==1)
        KernelGpuGradConvXAOnDevice<TYPE,3,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==4 && dimVect==1)
        KernelGpuGradConvXAOnDevice<TYPE,4,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==2)
        KernelGpuGradConvXAOnDevice<TYPE,2,2,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==3)
        KernelGpuGradConvXAOnDevice<TYPE,3,3,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==4 && dimVect==4)
        KernelGpuGradConvXAOnDevice<TYPE,4,4,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else {
        printf("GaussGpuGradConvXA error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_gradconv_xa.cu file !");
        hipFree(e_d);
        hipFree(alpha_d);
        hipFree(x_d);
        hipFree(y_d);
        hipFree(beta_d);
        hipFree(gamma_d);
        return(-1);
    }

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*dimVect),hipMemcpyDeviceToHost); // Output: N-by-E (same as alpha)

    // Free memory.
    hipFree(e_d);
    hipFree(alpha_d);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}

// Couldn't find a clean way to give a name to an explicit instantiation :-(

extern "C" int GaussGpuGradConvXA(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<__TYPE__,GaussFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int CauchyGpuGradConvXA(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<__TYPE__,CauchyFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int LaplaceGpuGradConvXA(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<__TYPE__,LaplaceFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int InverseMultiquadricGpuGradConvXA(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<__TYPE__,InverseMultiquadricFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}

void ExitFcn(void) {
    hipDeviceReset();
}
