/*
*	This cuda routine allows one to compute the derivative wrt the point cloud 'x' of the derivative
*	wrt 'x' of the expression
*		K(x_i,y_j) @ b_j =  sum_j f( |x_i-y_j|^2 ) b_j
*
*
*	We're looking for the gradient with respect to x of
*
*	< e, K(s,a,x,y,b) >  =  \sum_{i,j} f_s'( |x_i-y_j|^2 ) * < a_i, b_j > * 2 < e_i, x_i-y_j>,
*
*	which is an N-by-D array g_i (i from 1 to N), where each line is equal to
*
*	g_i  =  2* \sum_j < a_i, b_j > * [                       f_s'(  |x_i-y_j|^2 ) * e_i
*                                    + 2* < x_i-y_j, e_i > * f_s''( |x_i-y_j|^2 ) * (x_i-y_j) ]
*
*	We will compute this sum over the index 'j' on the GPU, with 'one thread' = 'one index i'.
*	Data will be stored as follow:
*	  - e_i in the thread memory
* 	  - a_i in the thread memory
*	  - x_i in the thread memory
*	  - y_j in the SharedData
*	  - b_j in the SharedData (beta_j, really)
*
*
* Author : Jean Feydy, heavily based on the work of Joan Glaunès and Benjamin Charlier.
*
*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "specific/radial_kernels/radial_kernels.h"
#include "specific/radial_kernels/cuda_gradconv_xx.cx"


//////////////////////////////////////////////////////
/////////// CPU -> GPU -> CPU routines ///////////////
//////////////////////////////////////////////////////


template < typename TYPE, KernelFun KernelFp , KernelFun KernelFpp >
int KernelGpuGradConvXX(TYPE ooSigma2,               // 1 / sigma^2
                        TYPE* e_h,                     // N-by-D array (same as x)
                        TYPE* alpha_h, TYPE* x_h,     // N-by-E, N-by-D arrays
                        TYPE* y_h,     TYPE* beta_h,  // M-by-D, M-by-E arrays
                        TYPE* gamma_h,                 // Output: N-by-D (same as x)
                        int dimPoint, int dimVect, int nx, int ny) { // D, E, N, M

    // Data on the device.
    TYPE* e_d;
    TYPE* alpha_d;
    TYPE* x_d;
    TYPE* y_d;
    TYPE* beta_d;
    TYPE* gamma_d;

    // Allocate arrays on device.
    hipMalloc((void**)&e_d,     sizeof(TYPE)*(nx*dimPoint));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*dimVect ));
    hipMalloc((void**)&x_d,     sizeof(TYPE)*(nx*dimPoint));
    hipMalloc((void**)&y_d,     sizeof(TYPE)*(ny*dimPoint));
    hipMalloc((void**)&beta_d,  sizeof(TYPE)*(ny*dimVect ));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*dimPoint)); // Output: N-by-D (same as x)

    // Send data from host to device.
    hipMemcpy(e_d,     e_h,     sizeof(TYPE)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*dimVect ), hipMemcpyHostToDevice);
    hipMemcpy(x_d,     x_h,     sizeof(TYPE)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(y_d,     y_h,     sizeof(TYPE)*(ny*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(beta_d,  beta_h,  sizeof(TYPE)*(ny*dimVect ), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    // Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time :
    if(     dimPoint==1 && dimVect==1)
        KernelGpuGradConvXXOnDevice<TYPE,1,1,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==1)
        KernelGpuGradConvXXOnDevice<TYPE,2,1,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==1)
        KernelGpuGradConvXXOnDevice<TYPE,3,1,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==4 && dimVect==1)
        KernelGpuGradConvXXOnDevice<TYPE,4,1,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==2)
        KernelGpuGradConvXXOnDevice<TYPE,2,2,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==3)
        KernelGpuGradConvXXOnDevice<TYPE,3,3,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==4 && dimVect==4)
        KernelGpuGradConvXXOnDevice<TYPE,4,4,KernelFp,KernelFpp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else {
        printf("GaussGpuGradConvXX error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_gradconv_xx.cu file !");
        hipFree(e_d);
        hipFree(alpha_d);
        hipFree(x_d);
        hipFree(y_d);
        hipFree(beta_d);
        hipFree(gamma_d);
        return(-1);
    }

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*dimPoint),hipMemcpyDeviceToHost); // Output: N-by-D (same as x)

    // Free memory.
    hipFree(e_d);
    hipFree(alpha_d);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}


// Couldn't find a clean way to give a name to an explicit instantiation :-(
extern "C" int GaussGpuGradConvXX(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXX<__TYPE__,GaussFp,GaussFpp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int CauchyGpuGradConvXX(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXX<__TYPE__,CauchyFp,CauchyFpp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int LaplaceGpuGradConvXX(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXX<__TYPE__,LaplaceFp,LaplaceFpp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int InverseMultiquadricGpuGradConvXX(__TYPE__ ooSigma2, __TYPE__* e_h, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXX<__TYPE__,InverseMultiquadricFp,InverseMultiquadricFpp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}

void ExitFcn(void) {
    hipDeviceReset();
}
