#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "specific/radial_kernels/radial_kernels.h"
#include "specific/radial_kernels/cuda_grad1conv.cx"


//////////////////////////////////////////////////////
/////////// CPU -> GPU -> CPU routines ///////////////
//////////////////////////////////////////////////////

template < typename TYPE, KernelFun KernelFp >
int KernelGpuGrad1Conv(TYPE ooSigma2,
                       TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h,
                       int dimPoint, int dimVect, int nx, int ny) {

    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* alpha_d;
    TYPE* gamma_d;
    TYPE* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d,     sizeof(TYPE)*(nx*dimPoint));
    hipMalloc((void**)&y_d,     sizeof(TYPE)*(ny*dimPoint));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*dimVect ));
    hipMalloc((void**)&beta_d,  sizeof(TYPE)*(ny*dimVect ));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*dimPoint));

    // Send data from host to device.
    hipMemcpy(x_d,     x_h,     sizeof(TYPE)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(y_d,     y_h,     sizeof(TYPE)*(ny*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*dimVect ), hipMemcpyHostToDevice);
    hipMemcpy(beta_d,  beta_h,  sizeof(TYPE)*(ny*dimVect ), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    // Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time :
    if(     dimPoint==1 && dimVect==1)
        KernelGpuGrad1ConvOnDevice<TYPE,1,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==1)
        KernelGpuGrad1ConvOnDevice<TYPE,2,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==1)
        KernelGpuGrad1ConvOnDevice<TYPE,3,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==4 && dimVect==1)
        KernelGpuGrad1ConvOnDevice<TYPE,4,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==2)
        KernelGpuGrad1ConvOnDevice<TYPE,2,2,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==3)
        KernelGpuGrad1ConvOnDevice<TYPE,3,3,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==4)
        KernelGpuGrad1ConvOnDevice<TYPE,4,4,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==4 && dimVect==4)
        KernelGpuGrad1ConvOnDevice<TYPE,4,4,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(TYPE)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else {
        printf("GaussGpuGrad1Conv error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_grad1conv.cu file !");
        hipFree(x_d);
        hipFree(y_d);
        hipFree(alpha_d);
        hipFree(gamma_d);
        hipFree(beta_d);
        return(-1);
    }

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*dimPoint),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);

    return 0;
}

// Couldn't find a clean way to give a name to an explicit instantiation :-(


// This instantiation bypass the function KernelGpuEvalConv as the pointers contain a address directly on the device
/*
 *extern "C" int GaussGpuEval_onDevice(__TYPE__ ooSigma2, __TYPE__* alpha_d, __TYPE__* x_d, __TYPE__* y_d, __TYPE__* beta_d, __TYPE__* gamma_d, int dimPoint, int dimVect, int nx, int ny) {
 *    dim3 blockSize (CUDA_BLOCK_SIZE,1,1); // number of threads in each block
 *    dim3 gridSize (nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1));
 *
 *    KernelGpuGrad1Conv<__TYPE__,3,3,GaussFp><<<gridSize,blockSize,blockSize.x*(3+3)*sizeof(__TYPE__)>>>
 *    (ooSigma2, x_d, y_d, beta_d, gamma_d, nx, ny);
 *    return 0;
 *}
 */
extern "C" int GaussGpuEval(__TYPE__ ooSigma2, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGrad1Conv<__TYPE__,GaussFp>(ooSigma2, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int LaplaceGpuEval(__TYPE__ ooSigma2, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGrad1Conv<__TYPE__,LaplaceFp>(ooSigma2, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int InverseMultiquadricGpuEval(__TYPE__ ooSigma2, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGrad1Conv<__TYPE__,InverseMultiquadricFp>(ooSigma2, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int CauchyGpuEval(__TYPE__ ooSigma2, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGrad1Conv<__TYPE__,CauchyFp>(ooSigma2, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}



void ExitFcn(void) {
    hipDeviceReset();
}
