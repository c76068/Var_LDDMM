#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "core/Pack.h"
#include "core/reductions/sum.h"
#include "core/reductions/log_sum_exp.h"

namespace keops {
template < typename TYPE, class FUN >
__global__ void GpuConv1DOnDevice(FUN fun, int nx, int ny, TYPE** px, TYPE** py, TYPE** pp) {

    // get the index of the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // declare shared mem
    extern __shared__ TYPE yj[];

    // get templated dimensions :
    typedef typename FUN::DIMSX DIMSX;  // DIMSX is a "vector" of templates giving dimensions of xi variables
    typedef typename FUN::DIMSY DIMSY;  // DIMSY is a "vector" of templates giving dimensions of yj variables
    typedef typename FUN::DIMSP DIMSP;  // DIMSP is a "vector" of templates giving dimensions of parameters variables
    const int DIMX = DIMSX::SUM;        // DIMX  is sum of dimensions for xi variables
    const int DIMY = DIMSY::SUM;        // DIMY  is sum of dimensions for yj variables
    const int DIMP = DIMSP::SUM;        // DIMP  is sum of dimensions for parameters variables
    const int DIMX1 = DIMSX::FIRST;     // DIMX1 is dimension of output variable

    // load parameter(s)
    TYPE param_loc[DIMP < 1 ? 1 : DIMP];
	load<DIMSP>(0,param_loc,pp); // load parameters variables from global memory to local thread memory

    // get the value of variable (index with i)
    TYPE xi[DIMX < 1 ? 1 : DIMX] ,tmp[DIMX1];
    if(i<nx) {
        InitializeOutput<TYPE,DIMX1,typename FUN::FORM>()(tmp); // tmp = 0
        load<DIMSX::NEXT>(i,xi+DIMX1,px+1); // load xi variables from global memory to local thread memory
    }

    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) {

        // get the current column
        int j = tile * blockDim.x + threadIdx.x;

        if(j<ny) { // we load yj from device global memory only if j<ny
            load<DIMSY>(j,yj+threadIdx.x*DIMY,py); // load yj variables from global memory to shared memory
        }
        __syncthreads();

        if(i<nx) { // we compute x1i only if needed
            TYPE* yjrel = yj; // Loop on the columns of the current block.
            for(int jrel = 0; (jrel < blockDim.x) && (jrel<ny-jstart); jrel++, yjrel+=DIMY) {
                call<DIMSX,DIMSY,DIMSP>(fun,xi,yjrel,param_loc); // Call the function, which accumulates results in xi[0:DIMX1]
                ReducePair<TYPE,DIMX1,typename FUN::FORM>()(tmp, xi);     // tmp += xi
            }
        }
        __syncthreads();
    }
    if(i<nx) {
        for(int k=0; k<DIMX1; k++)
            (*px)[i*DIMX1+k] = tmp[k];
    }

}

template < typename TYPE, class FUN >
int GpuConv1D_FromHost(FUN fun, int nx, int ny, TYPE** px_h, TYPE** py_h, TYPE** pp_h) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;
    const int DIMP = DIMSP::SUM;
    const int DIMX1 = DIMSX::FIRST;
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // pointers to device data
    TYPE *x_d, *y_d, *param_d;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    hipMalloc((void**)&p_data, sizeof(TYPE*)*(SIZEI+SIZEJ+SIZEP)+sizeof(TYPE)*(DIMP+nx*DIMX+ny*DIMY));

    TYPE **p_data_a = (TYPE**)p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;
    p_data_a += SIZEP;
    TYPE *p_data_b = (TYPE*)p_data_a;
    param_d = p_data_b;
    p_data_b += DIMP;
    x_d = p_data_b;
    p_data_b += nx*DIMX;
    y_d = p_data_b;

    // host arrays of pointers to device data
    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    int nvals;    
    php_d[0] = param_d;
    nvals = DIMSP::VAL(0);
    // if DIMSP is empty (i.e. no parameter), nvals = -1 which could result in a segfault
    if(nvals >= 0){ 
        hipMemcpy(php_d[0], pp_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }
    for(int k=1; k<SIZEP; k++) {
        php_d[k] = php_d[k-1] + nvals;
        nvals = DIMSP::VAL(k);
        hipMemcpy(php_d[k], pp_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }    

    phx_d[0] = x_d;
    nvals = nx*DIMSX::VAL(0);
    for(int k=1; k<SIZEI; k++) {
        phx_d[k] = phx_d[k-1] + nvals;
        nvals = nx*DIMSX::VAL(k);
        hipMemcpy(phx_d[k], px_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }

    phy_d[0] = y_d;
    nvals = ny*DIMSY::VAL(0);
    hipMemcpy(phy_d[0], py_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    for(int k=1; k<SIZEJ; k++) {
        phy_d[k] = phy_d[k-1] + nvals;
        nvals = ny*DIMSY::VAL(k);
        hipMemcpy(phy_d[k], py_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }

    // copy arrays of pointers
    hipMemcpy(pp_d, php_d, SIZEP*sizeof(TYPE*), hipMemcpyHostToDevice);
    hipMemcpy(px_d, phx_d, SIZEI*sizeof(TYPE*), hipMemcpyHostToDevice);
    hipMemcpy(py_d, phy_d, SIZEJ*sizeof(TYPE*), hipMemcpyHostToDevice);

    // Compute on device : grid is 2d and block is 1d
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv1DOnDevice<TYPE><<<gridSize,blockSize,blockSize.x*(DIMY)*sizeof(TYPE)>>>(fun,nx,ny,px_d,py_d,pp_d);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(*px_h, x_d, sizeof(TYPE)*(nx*DIMX1),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(p_data);

    return 0;
}

template < typename TYPE, class FUN >
int GpuConv1D_FromDevice(FUN fun, int nx, int ny, TYPE** phx_d, TYPE** phy_d, TYPE** php_d) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int DIMY = DIMSY::SUM;
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    hipMalloc((void**)&p_data, sizeof(TYPE*)*(SIZEI+SIZEJ+SIZEP));

    TYPE **p_data_a = (TYPE**)p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;

    hipMemcpy(px_d, phx_d, SIZEI*sizeof(TYPE*), hipMemcpyHostToDevice);
    hipMemcpy(py_d, phy_d, SIZEJ*sizeof(TYPE*), hipMemcpyHostToDevice);
    hipMemcpy(pp_d, php_d, SIZEP*sizeof(TYPE*), hipMemcpyHostToDevice);

    // Compute on device : grid and block are both 1d
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv1DOnDevice<TYPE><<<gridSize,blockSize,blockSize.x*(DIMY)*sizeof(TYPE)>>>(fun,nx,ny,px_d,py_d,pp_d);

    // block until the device has completed
    hipDeviceSynchronize();

    hipFree(p_data);

    return 0;
}

// and use getlist to enroll them into "pointers arrays" px and py.
template < typename TYPE, class FUN, typename... Args >
int GpuConv1D(FUN fun, int nx, int ny, TYPE* x1_h, Args... args) {

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    getlist<INDSI>(px_h+1,args...);
    getlist<INDSJ>(py_h,args...);
    getlist<INDSP>(pp_h,args...);

    return GpuConv1D_FromHost(fun,nx,ny,px_h,py_h,pp_h);

}

// Idem, but with args given as an array of arrays, instead of an explicit list of arrays
template < typename TYPE, class FUN >
int GpuConv1D(FUN fun, int nx, int ny, TYPE* x1_h, TYPE** args) {
    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    for(int i=1; i<SIZEI; i++)
        px_h[i] = args[INDSI::VAL(i-1)];
    for(int i=0; i<SIZEJ; i++)
        py_h[i] = args[INDSJ::VAL(i)];
    for(int i=0; i<SIZEP; i++)
        pp_h[i] = args[INDSP::VAL(i)];

    return GpuConv1D_FromHost(fun,nx,ny,px_h,py_h,pp_h);

}

// Same wrappers, but for data located on the device
template < typename TYPE, class FUN, typename... Args >
int GpuConv1D_FromDevice(FUN fun, int nx, int ny, TYPE* x1_d, Args... args) {

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    phx_d[0] = x1_d;

    getlist<INDSI>(phx_d+1,args...);
    getlist<INDSJ>(phy_d,args...);
    getlist<INDSP>(php_d,args...);

    return GpuConv1D_FromDevice(fun,nx,ny,phx_d,phy_d,php_d);

}

template < typename TYPE, class FUN >
int GpuConv1D_FromDevice(FUN fun, int nx, int ny, TYPE* x1_d, TYPE** args) {
    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_d[SIZEI];
    TYPE *py_d[SIZEJ];
    TYPE *pp_d[SIZEP];

    px_d[0] = x1_d;
    for(int i=1; i<SIZEI; i++)
        px_d[i] = args[INDSI::VAL(i-1)];
    for(int i=0; i<SIZEJ; i++)
        py_d[i] = args[INDSJ::VAL(i)];
    for(int i=0; i<SIZEP; i++)
        pp_d[i] = args[INDSP::VAL(i)];

    return GpuConv1D_FromDevice(fun,nx,ny,px_d,py_d,pp_d);

}

}
