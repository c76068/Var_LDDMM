#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "core/Pack.h"
#include "core/reductions/sum.h"
#include "core/reductions/log_sum_exp.h"

namespace keops {
template <typename T>
__device__ static constexpr T static_max_device(T a, T b) {
    return a < b ? b : a;
}

template <typename TYPE, int DIMVECT, class FUN>
__global__ void reduce2D(TYPE* in, TYPE* out, int sizeY,int nx) {
    /* Function used as a final reduction pass in the 2D scheme,
     * once the block reductions have been made.
     * Takes as input:
     * - in,  a  sizeY * (nx * DIMVECT ) array
     * - out, an          nx * DIMVECT   array
     *
     * Computes, in parallel, the "columnwise"-sum (which correspond to lines of blocks)
     * of *in and stores the result in out.
     */
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    /* As shown below, the code that is used to store the block-wise sum
      "tmp" in parallel is:
        if(i<nx)
            for(int k=0; k<DIMX1; k++)
                (*px)[blockIdx.y*DIMX1*nx+i*DIMX1+k] = tmp[k];
    */

    /* // This code should be a bit more efficient (more parallel) in the case
       // of a simple "fully parallel" reduction op such as "sum", "max" or "min"
    TYPE res = 0;
    if(tid < nx*DIMVECT) {
        for (int i = 0; i < sizeY; i++)
            res += in[tid + i*nx*DIMVECT]; // We use "+=" as a reduction op. But it could be anything, really!
        // res = in[tid+ nx* DIMVECT];
        out[tid] = res;
    }
    */

    // However, for now, we use a "vectorized" reduction op.,
    // which can also handle non-trivial reductions such as "LogSumExp"
    TYPE res[DIMVECT];
    InitializeOutput<TYPE,DIMVECT,typename FUN::FORM>()(res); // res = 0
    if(tid < nx) {
        for (int y = 0; y < sizeY; y++)
            ReducePair<TYPE,DIMVECT,typename FUN::FORM>()(res, in + (tid+y*nx)*DIMVECT); // res += in[(tid+y*nx) *DIMVECT : +DIMVECT];
        for (int k = 0; k < DIMVECT; k++) // copy to output
            out[tid*DIMVECT+k] = res[k];
    }

}

// thread kernel: computation of x1i = sum_j k(x2i,x3i,...,y1j,y2j,...) for index i given by thread id.
// N.B.: This routine by itself is generic, and does not specifically refer to the "sum" operation.
//       It can be used for any Map-Reduce operation, provided that "fun" is well-understood.
template < typename TYPE, class FUN >
__global__ void GpuConv2DOnDevice(FUN fun, int nx, int ny, TYPE** px, TYPE** py, TYPE** pp) {
    /*
     * px, py and pp are pointers to the device global memory.
     * They are arrays of arrays with the relevant size: for instance,
     * px[1] is a TYPE array of size ( nx * DIMSX::VAL(1) ).
     *
     * (*px) = px[0] is the output array, of size (nx * DIMSX::FIRST).
     *
     */
    // gets dimensions and number of variables of inputs of function FUN
    using DIMSX = typename FUN::DIMSX;  // DIMSX is a "vector" of templates giving dimensions of xi variables
    using DIMSY = typename FUN::DIMSY;  // DIMSY is a "vector" of templates giving dimensions of yj variables
    using DIMSP = typename FUN::DIMSP;  // DIMSP is a "vector" of templates giving dimensions of parameters variables
    const int DIMX = DIMSX::SUM;        // DIMX  is sum of dimensions for xi variables
    const int DIMY = DIMSY::SUM;        // DIMY  is sum of dimensions for yj variables
    const int DIMP = DIMSP::SUM;        // DIMP  is sum of dimensions for parameters variables
    const int DIMX1 = DIMSX::FIRST;     // DIMX1 is dimension of output variable

    // Load the parameter vector in the Thread Memory, for improved efficiency
    //TYPE param_loc[static_max_device(DIMP,1)];
    // (Jean :) Direct inlining to compile on Ubuntu 16.04 with nvcc7.5,
    //          which is a standard config in research. For whatever reason, I can't make
    //          it work an other way... Is it bad practice/performance?
    TYPE param_loc[DIMP < 1 ? 1 : DIMP];
	load<DIMSP>(0,param_loc,pp); // load parameters variables from global memory to local thread memory
    
    // Weird syntax to create a pointer in shared memory.
    extern __shared__ char yj_char[];
    TYPE* const yj = reinterpret_cast<TYPE*>(yj_char);

    // Step 1 : Load in Thread Memory the information needed in the current line ---------------------------
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    TYPE xi[DIMX];
    TYPE tmp[DIMX1];
    if(i<nx) { // we will compute x1i only if i is in the range
        InitializeOutput<TYPE,DIMX1,typename FUN::FORM>()(tmp); // tmp = 0
        // Load xi from device global memory.
        // Remember that we use an interleaved memory scheme where
        // xi = [ x1i, x2i, x3i, ... ].
        // Since we do not want to erase x1i, and only load x2i, x3i, etc.,
        // we add a small offset to the pointer given as an argument to the loading routine,
        // and ask it to only load "DIMSX::NEXT" bits of memory.
        load<DIMSX::NEXT>(i,xi+DIMX1,px+1); // load xi variables from global memory to local thread memory
    }

    // Step 2 : Load in Shared Memory the information needed in the current block of the product -----------
    // In the 1D scheme, we use a loop to run through the line.
    // In the 2D scheme presented here, the computation is done in parallel wrt both lines and columns.
    // Hence, we use "blockId.y" to get our current column number.
    int j = blockIdx.y * blockDim.x + threadIdx.x; // Same blockDim in x and y : squared tiles.
    if(j<ny) // we load yj from device global memory only if j<ny
        load<DIMSY>(j,yj+threadIdx.x*DIMY,py); // load yj variables from global memory to shared memory
    // More precisely : the j-th line of py is loaded to yj, at a location which depends on the
    // current threadId.

    __syncthreads(); // Make sure nobody lags behind

    // Step 3 : Once the data is loaded, execute fun --------------------------------------------------------
    // N.B.: There's no explicit summation here. Just calls to fun, which *accumulates* the results
    //       along the line, but does not *have* to use a "+=" as reduction operator.
    //       In the future, we could provide other reductions: max, min, ... whatever's needed.

    if(i<nx) { // we compute x1i only if needed
        TYPE* yjrel = yj; // Loop on the columns of the current block.
        for(int jrel = 0; (jrel<blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< ny); jrel++, yjrel+=DIMY) {
            call<DIMSX,DIMSY,DIMSP>(fun,xi,yjrel,param_loc); // Call the function, which accumulates results in xi[0:DIMX1]
            ReducePair<TYPE,DIMX1,typename FUN::FORM>()(tmp, xi);       // tmp += xi
        }
    }
    __syncthreads();

    // Step 4 : Save the result in global memory -----------------------------------------------------------
    // The current thread has computed the "linewise-sum" of a small block of the full Kernel Product
    // matrix, which corresponds to KP[ blockIdx.x * blockDim.x : (blockIdx.x+1) * blockDim.x ,
    //                                  blockIdx.y * blockDim.x : (blockIdx.y+1) * blockDim.x ]
    // We accumulate it in the output array (*px) = px[0], which has in fact gridSize.y * nx
    // lines of size DIMX1. The final reduction, which "sums over the block lines",
    // shall be done in a later step.
    if(i<nx)
        for(int k=0; k<DIMX1; k++)
            (*px)[blockIdx.y*DIMX1*nx+i*DIMX1+k] = tmp[k];
}
///////////////////////////////////////////////////


template < typename TYPE, class FUN >
int GpuConv2D_FromHost(FUN fun, int nx, int ny, TYPE** px_h, TYPE** py_h, TYPE** pp_h) {

    using DIMSX = typename FUN::DIMSX;
    using DIMSY = typename FUN::DIMSY;
    using DIMSP = typename FUN::DIMSP;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;
    const int DIMP = DIMSP::SUM;
    const int DIMX1 = DIMSX::FIRST;
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // Compute on device : grid is 2d and block is 1d
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  ny / blockSize.x + (ny%blockSize.x==0 ? 0 : 1);

    // Reduce  : grid and block are both 1d
    dim3 blockSize2;
    blockSize2.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMX1) / blockSize2.x + ((nx*DIMX1)%blockSize2.x==0 ? 0 : 1);


    // Data on the device. We need an "inflated" x1B, which contains gridSize.y "copies" of x_d
    // that will be reduced in the final pass.
    TYPE *x1B, *x_d, *y_d, *param_d;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    hipMalloc((void**)&p_data, sizeof(TYPE*)*(SIZEI+SIZEJ+SIZEP)+sizeof(TYPE)*(DIMP+nx*DIMX+ny*DIMY+nx*DIMX1*gridSize.y));

    TYPE **p_data_a = (TYPE**)p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;
    p_data_a += SIZEP;
    TYPE *p_data_b = (TYPE*)p_data_a;
    param_d = p_data_b;
    p_data_b += DIMP;
    x_d = p_data_b;
    p_data_b += nx*DIMX;
    y_d = p_data_b;
    p_data_b += ny*DIMY;
    x1B = p_data_b;

    // host arrays of pointers to device data
    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    // Send data from host to device.
    int nvals;
    php_d[0] = param_d;
    nvals = DIMSP::VAL(0);
    // if DIMSP is empty (i.e. no parameter), nvals = -1 which could result in a segfault
    if(nvals >= 0){ 
        hipMemcpy(php_d[0], pp_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }
    for(int k=1; k<SIZEP; k++) {
        php_d[k] = php_d[k-1] + nvals;
        nvals = DIMSP::VAL(k);
        hipMemcpy(php_d[k], pp_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }
    phx_d[0] = x_d;
    nvals = nx*DIMSX::VAL(0);
    for(int k=1; k<SIZEI; k++) {
        phx_d[k] = phx_d[k-1] + nvals;
        nvals = nx*DIMSX::VAL(k);
        hipMemcpy(phx_d[k], px_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }
    phy_d[0] = y_d;
    nvals = ny*DIMSY::VAL(0);
    hipMemcpy(phy_d[0], py_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    for(int k=1; k<SIZEJ; k++) {
        phy_d[k] = phy_d[k-1] + nvals;
        nvals = ny*DIMSY::VAL(k);
        hipMemcpy(phy_d[k], py_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }

    phx_d[0] = x1B; // we write the result before reduction in the "inflated" vector

    // copy arrays of pointers
    hipMemcpy(px_d, phx_d, SIZEI*sizeof(TYPE*), hipMemcpyHostToDevice);
    hipMemcpy(py_d, phy_d, SIZEJ*sizeof(TYPE*), hipMemcpyHostToDevice);
    hipMemcpy(pp_d, php_d, SIZEP*sizeof(TYPE*), hipMemcpyHostToDevice);

    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv2DOnDevice<TYPE><<<gridSize,blockSize,blockSize.x*(DIMY)*sizeof(TYPE)>>>(fun,nx,ny,px_d,py_d,pp_d);

    // Since we've used a 2D scheme, there's still a "blockwise" line reduction to make on
    // the output array px_d[0] = x1B. We go from shape ( gridSize.y * nx, DIMX1 ) to (nx, DIMX1)
    reduce2D<TYPE,DIMX1,FUN><<<gridSize2, blockSize2>>>(x1B, x_d, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(*px_h, x_d, sizeof(TYPE)*(nx*DIMX1),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(p_data);

    return 0;
}



template < typename TYPE, class FUN >
int GpuConv2D_FromDevice(FUN fun, int nx, int ny, TYPE** phx_d, TYPE** phy_d, TYPE** php_d) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int DIMY = DIMSY::SUM;
    const int DIMX1 = DIMSX::FIRST;
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // Data on the device. We need an "inflated" x1B, which contains gridSize.y "copies" of x_d
    // that will be reduced in the final pass.
    TYPE *x1B, *out;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // Compute on device : grid is 2d and block is 1d
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  ny / blockSize.x + (ny%blockSize.x==0 ? 0 : 1);

    // Reduce : grid and block are both 1d
    dim3 blockSize2;
    blockSize2.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMX1) / blockSize2.x + ((nx*DIMX1)%blockSize2.x==0 ? 0 : 1);

    // single hipMalloc
    void **p_data;

	hipMalloc((void**)&p_data, sizeof(TYPE*)*(SIZEI+SIZEJ+SIZEP)+sizeof(TYPE)*(nx*DIMX1*gridSize.y));

    TYPE **p_data_a = (TYPE**)p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;
    p_data_a += SIZEP;
    x1B = (TYPE*)p_data_a;

    out = phx_d[0]; // save the output location

    phx_d[0] = x1B;

	hipMemcpy(px_d, phx_d, SIZEI*sizeof(TYPE*), hipMemcpyHostToDevice);
	hipMemcpy(py_d, phy_d, SIZEJ*sizeof(TYPE*), hipMemcpyHostToDevice);
	hipMemcpy(pp_d, php_d, SIZEP*sizeof(TYPE*), hipMemcpyHostToDevice);

    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv2DOnDevice<TYPE><<<gridSize,blockSize,blockSize.x*(DIMY)*sizeof(TYPE)>>>(fun,nx,ny,px_d,py_d,pp_d);

    // Since we've used a 2D scheme, there's still a "blockwise" line reduction to make on
    // the output array px_d[0] = x1B. We go from shape ( gridSize.y * nx, DIMX1 ) to (nx, DIMX1)
    reduce2D<TYPE,DIMX1,FUN><<<gridSize2, blockSize2>>>(x1B, out, gridSize.y,nx);

    // block until the device has completed
    hipDeviceSynchronize();

    hipFree(p_data);

    return 0;
}


// Wrapper around GpuConv2D, which takes lists of arrays *x1, *x2, ..., *y1, *y2, ...
// and use getlist to enroll them into "pointers arrays" px and py.
template < typename TYPE, class FUN, typename... Args >
int GpuConv2D(FUN fun, int nx, int ny, TYPE* x1_h, Args... args) {

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    getlist<INDSI>(px_h+1,args...);
    getlist<INDSJ>(py_h,args...);
    getlist<INDSP>(pp_h,args...);

    return GpuConv2D_FromHost(fun,nx,ny,px_h,py_h,pp_h);

}

// Idem, but with args given as an array of arrays, instead of an explicit list of arrays
template < typename TYPE, class FUN >
int GpuConv2D(FUN fun, int nx, int ny, TYPE* x1_h, TYPE** args) {
    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    for(int i=1; i<SIZEI; i++)
        px_h[i] = args[INDSI::VAL(i-1)];
    for(int i=0; i<SIZEJ; i++)
        py_h[i] = args[INDSJ::VAL(i)];
    for(int i=0; i<SIZEP; i++)
        pp_h[i] = args[INDSP::VAL(i)];

    return GpuConv2D_FromHost(fun,nx,ny,px_h,py_h,pp_h);

}


// Same wrappers, but for data located on the device
template < typename TYPE, class FUN, typename... Args >
int GpuConv2D_FromDevice(FUN fun, int nx, int ny, TYPE* x1_d, Args... args) {

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_d[SIZEI];
    TYPE *py_d[SIZEJ];
    TYPE *pp_d[SIZEP];

    px_d[0] = x1_d;
    getlist<INDSI>(px_d+1,args...);
    getlist<INDSJ>(py_d,args...);
    getlist<INDSP>(pp_d,args...);

    return GpuConv2D_FromDevice(fun,nx,ny,px_d,py_d,pp_d);
}

template < typename TYPE, class FUN >
int GpuConv2D_FromDevice(FUN fun, int nx, int ny, TYPE* x1_d, TYPE** args) {
    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    phx_d[0] = x1_d;
    for(int i=1; i<SIZEI; i++)
        phx_d[i] = args[INDSI::VAL(i-1)];
    for(int i=0; i<SIZEJ; i++)
        phy_d[i] = args[INDSJ::VAL(i)];
    for(int i=0; i<SIZEP; i++)
        php_d[i] = args[INDSP::VAL(i)];

    return GpuConv2D_FromDevice(fun,nx,ny,phx_d,phy_d,php_d);

}



}
